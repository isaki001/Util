#include "hip/hip_runtime.h"
#include <stdio.h>
#include "util.cuh"

using namespace std;
 
 __global__ void hello_world(int *x){
   printf("hello from gpu\n");
   for(int i=0; i<10; i++)
    printf("%i\t", x[i]);
   printf("\n");
}

int main(){
  
  hipDeviceReset();
  int *dx, *dx2, *dx3;
  
  int x[10]  = {1,  2,  3,  4,  5,  6,  7,  8,  9,  10};
  int x2[10] = {-1, -2, -3, -4, -5, -6, -7, -8, -9, -10};
  int x3[10] = {-11, -22, -3, -4, -5, -6, -7, -8, -9, -10};
  
  hipMalloc((void **)&dx, sizeof(int)*10);
  hipMemcpy(dx, x, sizeof(int)*10, hipMemcpyHostToDevice);
  
  hipMalloc((void **)&dx2, sizeof(int)*10);
  hipMemcpy(dx2, x2, sizeof(int)*10, hipMemcpyHostToDevice);
  
  hipMalloc((void **)&dx3, sizeof(int)*10);
  hipMemcpy(dx3, x3, sizeof(int)*10, hipMemcpyHostToDevice);
  
  constexpr size_t numGPUArrays = 3;
  size_t gpuArraySize = 10;
  display<int, numGPUArrays>(gpuArraySize, dx, dx2, dx3);

  return 0;
}












